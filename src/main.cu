#include "hip/hip_runtime.h"
#include "utils/image.hpp"
#include <math.h>
#include <iostream>
#include "histogramCPU.hpp"
#include "histogramGPU.hpp"

#define OUT_CPU_IMAGE_PATH "./images/gpu-result.jpg"
#define OUT_GPU_IMAGE_PATH "./images/cpu-result.jpg"

int main(int argc, char const *argv[])
{
    int N = 4;
    int blockSize = 512;
    std::string path("./images/batiment-1.jpg");

    if(argc <= 1)
    {
        std::cout << "Missing argument !" << std::endl;
    }
    for (int i = 1; i < argc; ++i)
    {
        if ( !strcmp(argv[i], "-N"))
        {
            if (sscanf(argv[++i], "%i", &N) != 1)
                std::cout << "Error argument N" << std::endl;
        }
        else if ( !strcmp(argv[i], "-b"))
        {
            if (sscanf(argv[++i], "%i", &blockSize) != 1)
                std::cout << "Error argument b" << std::endl;
        }
        else if ( !strcmp(argv[i], "-i"))
        {
            path = std::string(argv[++i]);
            std::cout << "Image loaded -> " << path << std::endl;
        }
    }

    // Get the image width and height
    Image image;
    image.load(path.c_str());

    // Results
    const unsigned int imageSize = image._width * image._height;
    int * outCPU = new int[imageSize];
    int * outGPU = new int[imageSize];

    // CPU sequential
	std::cout << "============================================"	<< std::endl;
	std::cout << "         Sequential version on CPU          "	<< std::endl;
	std::cout << "============================================"	<< std::endl;

    float cpuTime = 0.f;
    HistogramCPU histogramCPU(path.c_str());
    cpuTime = histogramCPU.histogramEqualisation(OUT_CPU_IMAGE_PATH, outCPU);
    std::cout << "Time : " << cpuTime << std::endl;
    
    // GPU CUDA
	std::cout << "============================================"	<< std::endl;
	std::cout << "         Parallel version on GPU            "	<< std::endl;
	std::cout << "============================================"	<< std::endl;

    float gpuTime = 0.f;
    HistogramGPU histogramGPU(path.c_str());
    gpuTime = histogramGPU.histogramEqualisation(OUT_GPU_IMAGE_PATH, outGPU, N, blockSize);
    std::cout << "Time : " << gpuTime << std::endl;

    // CHECK RESULT
    std::cout << "============================================"	<< std::endl;
	std::cout << "              Checking results		      "	<< std::endl;
	std::cout << "============================================"	<< std::endl;

	for ( int i = 0; i < imageSize; ++i ) 
	{
        // Result may be slightly different between CPU and GPU because of the floating-point calculation
        if ( abs(outCPU[i] - outGPU[i]) > 2)  { 
            std::cerr << "Error for index " << i <<" :" << std::endl;
            std::cerr << "\t CPU: [" << outCPU[i] << "]" << std::endl;
            std::cerr << "\t GPU: [" << outGPU[i] << "]" << std::endl;
            exit( EXIT_FAILURE );
        }
	}
	std::cout << "Congratulations! Job's done!" << std::endl << std::endl;

    // FREE MEMORY
    delete[] outCPU;
    delete[] outGPU;

    return 0;
}
